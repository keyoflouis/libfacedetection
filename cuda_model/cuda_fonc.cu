#include "cuda_fonc.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define thread_per_block 256

CDataBlobKernel *convolution_1x1pointwiseKernel(int input_rows,
                                                int input_cols,
                                                int input_channels,
                                                int input_channelStep,
                                                float *input_data,

                                                int channels,
                                                int num_filters,
                                                bool is_depthwise,
                                                bool is_pointwise,
                                                bool with_relu,

                                                int weights_rows,
                                                int weights_cols,
                                                int weights_channels,
                                                int weights_channelStep,
                                                float *weight_data,

                                                int biases_rows,
                                                int biases_cols,
                                                int biases_channels,
                                                int biases_channelStep,
                                                float *biases_data,

                                                int output_rows,
                                                int output_cols,
                                                int output_channels,
                                                int output_channelStep,
                                                float *output_data){
	
	CDataBlobKernel inputData(input_rows, input_cols, input_channels, input_channelStep, input_data);
	CDataBlobKernel outputData(output_rows, output_cols, output_channels, output_channelStep, output_data);
	FiltersKernel filters(  channels, num_filters, is_depthwise, is_pointwise, with_relu,
                      		weights_rows, weights_cols, weights_channels, weights_channelStep, weight_data,
                      		biases_rows, biases_cols, biases_channels, biases_channelStep, biases_data);



	
	CDataBlobKernel* dev_inputData;
	CDataBlobKernel* dev_outputData;
	FiltersKernel* dev_filters;


	hipMallocManaged((void**)&dev_inputData,sizeof(CDataBlobKernel));
	hipMallocManaged((void**)&dev_outputData,sizeof(CDataBlobKernel));
	hipMallocManaged((void**)&dev_filters,sizeof(FiltersKernel));
	
	
	float* dev_input_data;
	float* dev_output_data;

	// allocate and deep copy for inpute
 	size_t size_bytes_devInputData = size_t(inputData.rows) * inputData.cols * inputData.channelStep;
	hipMallocManaged((void**)&dev_input_data, size_bytes_devInputData);
	hipMemcpy(dev_input_data ,inputData.data, size_bytes_devInputData,hipMemcpyHostToDevice);
	hipMemcpy( &dev_inputData->data , &dev_input_data , sizeof(float*) , hipMemcpyHostToDevice );

	// allocate and deep copy for outpute
	size_t size_bytes_devOutputData = size_t(outputData.rows) * outputData.cols * outputData.channelStep;
	hipMallocManaged((void**)&dev_output_data , size_bytes_devOutputData );
	hipMemcpy(dev_output_data, outputData.data, size_bytes_devOutputData, hipMemcpyHostToDevice);
	hipMemcpy( &dev_outputData->data , &dev_output_data , sizeof(float*) , hipMemcpyHostToDevice );


	
	// allocate and deep copy for filters
	float* dev_filters_weightsData;
	float* dev_filters_biasesData;
	
	size_t size_bytes_devfilters_weightsData = size_t(filters.weights.rows) * filters.weights.cols * filters.weights.channelStep;
	size_t size_bytes_devfilters_biasesData = size_t(filters.biases.rows) * filters.biases.cols * filters.biases.channelStep;

	hipMallocManaged((void**)&dev_filters_weightsData,size_bytes_devfilters_weightsData);
	hipMallocManaged((void**)&dev_filters_biasesData,size_bytes_devfilters_biasesData);
	
	hipMemcpy(dev_filters_weightsData ,  filters.weights.data , size_bytes_devfilters_weightsData ,hipMemcpyHostToDevice);
	hipMemcpy(&dev_filters->weights.data,&dev_filters_weightsData, size_bytes_devfilters_weightsData,hipMemcpyHostToDevice);
	
	hipMemcpy(dev_filters_biasesData ,  filters.biases.data , size_bytes_devfilters_biasesData ,hipMemcpyHostToDevice);
	hipMemcpy(&dev_filters->biases.data,&dev_filters_biasesData, size_bytes_devfilters_biasesData,hipMemcpyHostToDevice);

	// invoke	


	// store the results
	hipMemcpy(&outputData , dev_outputData,sizeof(CDataBlobKernel),hipMemcpyDeviceToHost);
	hipMemcpy(&outputData.data , &dev_outputData->data,size_bytes_devOutputData ,hipMemcpyDeviceToHost);

	// free
	
	hipFree(dev_input_data);
	hipFree(dev_inputData->data);
	hipFree(dev_inputData);

	hipFree(dev_output_data);
	hipFree(dev_outputData->data);
	hipFree(dev_outputData);

	hipFree(dev_filters_weightsData);
	hipFree(dev_filters_biasesData);
	hipFree(dev_filters->biases.data);
	hipFree(dev_filters->weights.data);
	hipFree(dev_filters);

    for (int row = 0; row < outputData.rows; row++)
    {
        for (int col = 0; col < outputData.cols; col++)
        {
            float *pOut = outputData.ptr(row, col);
            const float *pIn = inputData.ptr(row, col);

            for (int ch = 0; ch < outputData.channels; ch++)
            {
                const float *pF = filters.weights.ptr(0, ch);
                float sum = 0.f;
                for (int i = 0; i < inputData.channels; i++)
                {
                    sum += (pIn[i] * pF[i]);
                }
                pOut[ch] = sum;
                pOut[ch] += filters.biases.data[ch];
            }
        }
    }
    return &outputData;
};