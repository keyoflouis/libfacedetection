#include "hip/hip_runtime.h"
#include "cuda_fonc.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define thread_per_block 256
__global__ void kernel(CDataBlobKernel* inputData, CDataBlobKernel* outputData, FiltersKernel* filters) {
	
	printf("inputData : %d , %d ,%d ,%d,%d \n", inputData->channels, inputData->channelStep, inputData->cols, inputData->data,inputData->rows);
	printf("outputData : %d , %d ,%d ,%d,%d \n", outputData->channels, outputData->channelStep, outputData->cols, outputData->data, outputData->rows);
}

CDataBlobKernel *convolution_1x1pointwiseKernel(int input_rows,
                                                int input_cols,
                                                int input_channels,
                                                int input_channelStep,
                                                float *input_data,

                                                int channels,
                                                int num_filters,
                                                bool is_depthwise,
                                                bool is_pointwise,
                                                bool with_relu,

                                                int weights_rows,
                                                int weights_cols,
                                                int weights_channels,
                                                int weights_channelStep,
                                                float *weight_data,

                                                int biases_rows,
                                                int biases_cols,
                                                int biases_channels,
                                                int biases_channelStep,
                                                float *biases_data,

                                                int output_rows,
                                                int output_cols,
                                                int output_channels,
                                                int output_channelStep,
                                                float *output_data){
	
	CDataBlobKernel inputData(input_rows, input_cols, input_channels, input_channelStep, input_data);
	CDataBlobKernel outputData(output_rows, output_cols, output_channels, output_channelStep, output_data);
	FiltersKernel filters(  channels, num_filters, is_depthwise, is_pointwise, with_relu,
                      		weights_rows, weights_cols, weights_channels, weights_channelStep, weight_data,
                      		biases_rows, biases_cols, biases_channels, biases_channelStep, biases_data);


	
	
	CDataBlobKernel* dev_inputData;
	CDataBlobKernel* dev_outputData;
	FiltersKernel* dev_filters;


	hipMalloc((void**)&dev_inputData,sizeof(CDataBlobKernel));
	hipMalloc((void**)&dev_outputData,sizeof(CDataBlobKernel));
	hipMalloc((void**)&dev_filters,sizeof(FiltersKernel));
	
	
	float* dev_input_data;
	float* dev_output_data;

	// allocate copy and deep copy for inpute
 	size_t size_bytes_devInputData = size_t(inputData.rows) * inputData.cols * inputData.channelStep;
	hipMemcpy(dev_inputData,&inputData,sizeof(CDataBlobKernel),hipMemcpyHostToDevice);


	// deep copy for inpute
	hipMalloc((void**)&dev_input_data, size_bytes_devInputData);
	hipMemcpy(dev_input_data ,inputData.data, size_bytes_devInputData,hipMemcpyHostToDevice);
	hipMemcpy( &dev_inputData->data , &dev_input_data , sizeof(float*) , hipMemcpyHostToDevice );

	// allocate copy and deep copy for outpute
	size_t size_bytes_devOutputData = size_t(outputData.rows) * outputData.cols * outputData.channelStep;
	hipMemcpy(dev_outputData, &outputData, sizeof(CDataBlobKernel), hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_output_data , size_bytes_devOutputData );
	hipMemcpy(dev_output_data, outputData.data, size_bytes_devOutputData, hipMemcpyHostToDevice);
	hipMemcpy( &dev_outputData->data , &dev_output_data , sizeof(float*) , hipMemcpyHostToDevice );


	
	// allocate and deep copy for filters
	float* dev_filters_weightsData;
	float* dev_filters_biasesData;
	hipMemcpy(dev_filters, &filters,sizeof(FiltersKernel),hipMemcpyHostToDevice);


	size_t size_bytes_devfilters_weightsData = size_t(filters.weights.rows) * filters.weights.cols * filters.weights.channelStep;
	size_t size_bytes_devfilters_biasesData = size_t(filters.biases.rows) * filters.biases.cols * filters.biases.channelStep;

	hipMalloc((void**)&dev_filters_weightsData,size_bytes_devfilters_weightsData);
	hipMalloc((void**)&dev_filters_biasesData,size_bytes_devfilters_biasesData);
	
	hipMemcpy(dev_filters_weightsData ,  filters.weights.data , size_bytes_devfilters_weightsData ,hipMemcpyHostToDevice);
	hipMemcpy(&dev_filters->weights.data,&dev_filters_weightsData, size_bytes_devfilters_weightsData,hipMemcpyHostToDevice);
	
	hipMemcpy(dev_filters_biasesData ,  filters.biases.data , size_bytes_devfilters_biasesData ,hipMemcpyHostToDevice);
	hipMemcpy(&dev_filters->biases.data,&dev_filters_biasesData, size_bytes_devfilters_biasesData,hipMemcpyHostToDevice);

	// invoke	
	kernel << <1, 1 >> > (dev_inputData,dev_outputData,dev_filters);

	// store the results
	void* temp = outputData.data;
	hipMemcpy(&outputData , dev_outputData,sizeof(CDataBlobKernel),hipMemcpyDeviceToHost);
	outputData.data = (float*)temp;
	hipMemcpy(outputData.data, dev_output_data, size_bytes_devOutputData, hipMemcpyDeviceToHost);


	// free
	
	hipFree(dev_input_data);
	hipFree(dev_inputData);

	hipFree(dev_output_data);
	hipFree(dev_outputData);

	hipFree(dev_filters_weightsData);
	hipFree(dev_filters_biasesData);
	hipFree(dev_filters);
	
    for (int row = 0; row < outputData.rows; row++)
    {
        for (int col = 0; col < outputData.cols; col++)
        {
            float *pOut = outputData.ptr(row, col);
            const float *pIn = inputData.ptr(row, col);

            for (int ch = 0; ch < outputData.channels; ch++)
            {
                const float *pF = filters.weights.ptr(0, ch);
                float sum = 0.f;
                for (int i = 0; i < inputData.channels; i++)
                {
                    sum += (pIn[i] * pF[i]);
                }
                pOut[ch] = sum;
                pOut[ch] += filters.biases.data[ch];
            }
        }
    }
    return &outputData;
};