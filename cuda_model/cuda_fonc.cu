#include "cuda_fonc.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define thread_per_block 256

CDataBlobKernel *convolution_1x1pointwiseKernel(int input_rows,
                                                int input_cols,
                                                int input_channels,
                                                int input_channelStep,
                                                float *input_data,

                                                int channels,
                                                int num_filters,
                                                bool is_depthwise,
                                                bool is_pointwise,
                                                bool with_relu,

                                                int weights_rows,
                                                int weights_cols,
                                                int weights_channels,
                                                int weights_channelStep,
                                                float *weight_data,

                                                int biases_rows,
                                                int biases_cols,
                                                int biases_channels,
                                                int biases_channelStep,
                                                float *biases_data,

                                                int output_rows,
                                                int output_cols,
                                                int output_channels,
                                                int output_channelStep,
                                                float *output_data)
{
    CDataBlobKernel inputData(input_rows, input_cols, input_channels, input_channelStep, input_data);
    CDataBlobKernel outputData(output_rows, output_cols, output_channels, output_channelStep, output_data);

    FiltersKernel filters(channels, num_filters, is_depthwise, is_pointwise, with_relu,
                          weights_rows, weights_cols, weights_channels, weights_channelStep, weight_data,
                          biases_rows, biases_cols, biases_channels, biases_channelStep, biases_data);
    
    CDataBlobKernel *host_inputData = &inputData;
    CDataBlobKernel *host_outputData = &outputData;
    FiltersKernel *host_filters = &filters;

    CDataBlobKernel *dev_inputData =nullptr;
    CDataBlobKernel *dev_outputData =nullptr;
    FiltersKernel *dev_filters =nullptr;

    hipMallocManaged((void **)&dev_inputData, sizeof(CDataBlobKernel));
    hipMallocManaged((void **)&dev_outputData, sizeof(CDataBlobKernel));
    hipMallocManaged((void **)&dev_filters, sizeof(FiltersKernel));

    // define the device pointer
    hipMemcpy((void **)&dev_inputData, (void **)&host_inputData, sizeof(CDataBlobKernel), hipMemcpyHostToDevice);
    hipMemcpy((void **)&dev_outputData, (void **)&host_outputData, sizeof(CDataBlobKernel), hipMemcpyHostToDevice);
    hipMemcpy((void **)&dev_filters, (void **)&host_filters, sizeof(FiltersKernel), hipMemcpyHostToDevice);

    //  inite devices pointers
    //  dev_inputData
    size_t size_bytes_devInputData = size_t(host_inputData->rows) * host_inputData->cols * host_inputData->channelStep;
    hipMallocManaged((void **)(&dev_inputData->data), size_bytes_devInputData);
    hipMemcpy((void **)(&dev_inputData->data), (void **)(&host_inputData->data), sizeof(size_bytes_devInputData), hipMemcpyHostToDevice);

    // dev_outputData
    size_t size_bytes_devOutputData = size_t(host_outputData->rows) * host_outputData->cols * host_outputData->channelStep;
    hipMallocManaged((void **)(&dev_outputData->data), sizeof(size_bytes_devOutputData));
    hipMemcpy((void **)(&dev_outputData->data), (void **)(&host_outputData->data), size_bytes_devOutputData, hipMemcpyHostToDevice);

    // dev_filters->weights.data
    size_t size_bytes_devfilters_weightsData = size_t(host_filters->weights.rows) * host_filters->weights.cols * host_filters->weights.channelStep;
    hipMallocManaged((void **)(&dev_filters->weights.data), size_bytes_devfilters_weightsData);
    hipMemcpy((void **)(&dev_filters->weights.data), (void **)(&host_filters->weights.data), size_bytes_devfilters_weightsData, hipMemcpyHostToDevice);

    // dev_filters->biases
    size_t size_bytes_devfilters_biasesData = size_t(host_filters->biases.rows) * host_filters->biases.cols * host_filters->biases.channelStep;
    hipMallocManaged((void **)(&dev_filters->biases.data), size_bytes_devfilters_biasesData);
    hipMemcpy((void **)(&dev_filters->biases.data), (void **)(&host_filters->biases.data), size_bytes_devfilters_biasesData, hipMemcpyHostToDevice);


    // kernnel


    // store the results into the host_outputData
    hipMemcpy((void **)&host_outputData, (void **)&dev_outputData, sizeof(CDataBlobKernel), hipMemcpyDeviceToHost);
    hipMemcpy((void **)(&host_outputData->data), (void **)(&dev_outputData->data), size_bytes_devOutputData, hipMemcpyDeviceToHost);

    // free the allocated memory
    hipFree(dev_inputData->data);
    hipFree(dev_outputData->data);
    hipFree(dev_filters->weights.data);
    hipFree(dev_filters->biases.data);

    hipFree(dev_inputData);
    hipFree(dev_outputData);
    hipFree(dev_filters);

    for (int row = 0; row < outputData.rows; row++)
    {
        for (int col = 0; col < outputData.cols; col++)
        {
            float *pOut = outputData.ptr(row, col);
            const float *pIn = inputData.ptr(row, col);

            for (int ch = 0; ch < outputData.channels; ch++)
            {
                const float *pF = filters.weights.ptr(0, ch);
                float sum = 0.f;
                for (int i = 0; i < inputData.channels; i++)
                {
                    sum += (pIn[i] * pF[i]);
                }
                pOut[ch] = sum;
                pOut[ch] += filters.biases.data[ch];
            }
        }
    }
    return &outputData;
};